
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

/*
 * In the current application, `N` is larger than the grid.
 * Refactor this kernel to use a grid-stride loop in order that
 * each parallel thread work on more than one element of the array.
 */

__global__
void doubleElements(int *a, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x; // stride를 계산해서 한 grid마다 여러 개의 데이터를 for문으로 작업하는 방법이 있다.
  int i = 0;
  for (i = idx; i < N; i += stride)
  {
    a[i] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  /*
   * `N` is greater than the size of the grid (see below).
   */

  int N = 10000;
  int *a;
  hipError_t error;
  size_t size = N * sizeof(int);
  error = hipMallocManaged(&a, size); // error 를 다룬다면 cudaMalloc의 경우 이 방법을 쓴다.
  if (error != hipSuccess)
  {
    printf("Error: %s\n", hipGetErrorString(error)); // `cudaGetErrorString` is provided by CUDA.
  }
  init(a, N);

  /*
   * The size of this grid is 256*32 = 8192.
   */

  size_t threads_per_block = 256;
  size_t number_of_blocks = 32;

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  hipDeviceSynchronize();

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}