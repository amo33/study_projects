// 2D 3D dimension of Grid and blocks
// Grids와 block들은 3차원까지 선언할 수 있다.
// 다차원으로 선언하는 것이 성능에 영향을 주는 것은 아니다.
// 허나, 2d 행렬과 같은 input data (특정: matrix들)을 다룰 때는 도움이 된다.
// 이 다차원을 선언하기 위해서 cuda에 있는 dim3 형으로 가능하다. 

/*
가령 아래와 같이 선언하면 block 마다 스레드, block의 개수도 dim3형태로 선언되어 사용될 수 있다.
dim3 threads_per_block(16, 16, 1); 
dim3 number_of_blocks(16, 16, 1);
*/
/*
명세서 

This code contains a host function matrixMulCPU which is fully functional.
Your task is to build out the matrixMulGPUCUDA kernel. 
The source code will execute the matrix multiplication with both functions, and compare their answers to verify the correctness of the CUDA kernel you will be writing. 

You will need to create an execution configuration whose arguments are both dim3 values with the x and y dimensions set to greater than 1.
Inside the body of the kernel, you will need to establish the running thread’s unique index within the grid per usual, but you should establish two indices for the thread: 
one for the x axis of the grid, and one for the y axis of the grid.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define N  64
#define ROW 16
#define COLUMN 16
__global__ void matrixMulGPU( int * a, int * b, int * c )
{
  /*
   * Build out this kernel.
   */
  int val = 0;
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  if (row < N && col <N)
  {
    for (int k=0; k < N; ++k)
    {
      val += a[row*N+k] * b[k*N+col];
    }
    c[row*N+col] = val;
  }
}

/*
 * This CPU function already works, and will run to create a solution matrix
 * against which to verify your work building out the matrixMulGPU kernel.
 */

void matrixMulCPU( int * a, int * b, int * c )
{
  int val = 0;

  for( int row = 0; row < N; ++row )
    for( int col = 0; col < N; ++col )
    {
      val = 0;
      for ( int k = 0; k < N; ++k )
        val += a[row * N + k] * b[k * N + col];
      c[row * N + col] = val;
    }
}

int main()
{
  int *a, *b, *c_cpu, *c_gpu; // Allocate a solution matrix for both the CPU and the GPU operations

  int size = N * N * sizeof (int); // Number of bytes of an N x N matrix

  // Allocate memory
  hipMallocManaged (&a, size);
  hipMallocManaged (&b, size);
  hipMallocManaged (&c_cpu, size);
  hipMallocManaged (&c_gpu, size);

  // Initialize memory; create 2D matrices
  for( int row = 0; row < N; ++row )
    for( int col = 0; col < N; ++col )
    {
      a[row*N + col] = row;
      b[row*N + col] = col+2;
      c_cpu[row*N + col] = 0;
      c_gpu[row*N + col] = 0;
    }

  /*
   * Assign `threads_per_block` and `number_of_blocks` 2D values
   * that can be used in matrixMulGPU above.
   */

  dim3 threads_per_block(ROW, COLUMN, 1);
  dim3 number_of_blocks ((N / threads_per_block.x) + 1, (N / threads_per_block.y) + 1, 1);
  
  matrixMulGPU <<< number_of_blocks, threads_per_block >>> ( a, b, c_gpu );

  hipDeviceSynchronize();

  // Call the CPU version to check our work
  matrixMulCPU( a, b, c_cpu );

  // Compare the two answers to make sure they are equal
  bool error = false;
  for( int row = 0; row < N && !error; ++row )
    for( int col = 0; col < N && !error; ++col )
      if (c_cpu[row * N + col] != c_gpu[row * N + col])
      {
        printf("FOUND ERROR at c[%d][%d]\n", row, col);
        error = true;
        break;
      }
  if (!error)
    printf("Success!\n");

  // Free all our allocated memory
  hipFree(a); hipFree(b);
  hipFree( c_cpu ); hipFree( c_gpu );
}