
#include <hip/hip_runtime.h>
#include <stdio.h>
/*

이 코드는 CPU로만 작동하는 벡터 더하기 어플리케이션이다. 여기 있는 addVectorsInto 함수를 CUDA kernel로 만들어 GPU 병렬 연산을 할 수 있게 만들어보자. 다음을 유의해서 코드를 짜보자.

addVectorsInto를 CUDA kernel로 만들기
addVectorsInto가 CUDA kernel로 작동하는 적절한 execution configuration을 찾고 실행하기
메모리 할당과 해제를 적절히 해서 a, b, result 벡터가 CPU/GPU에서 모두 접근 가능하도록 하기
addVectorsInto를 리팩토링하자: it will be launched inside of a single thread, and only needs to do one thread’s worth of work on the input vectors. Be certain the thread will never try to access elements outside the range of the input vectors, and take care to note whether or not the thread needs to do work on more than one element of the input vectors.
CUDA 코드가 잘못될 수 있는 부분에 적절히 error handling을 하자.

*/
// inline 이란 : ?????????????????????????????????????????????
// fprintf란: fprintf를 사용하면 문자열이 파일에 출력된다는 뜻이다.
// -> int fprintf(FILE *stream, const char* format, ...)
// stderr stream : 주로 에러 메시지를 출력하기 위해 만들어진 stream
inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess)
  {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x*blockDim.x;
  for(int i = idx; i < N; i+=stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20; // Don't change
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;
  hipError_t mallocCheck, syncError, asyncError;
  size_t threads_per_block = 1024; // block에 존재할 수 있는 thread의 개수는 최대 1024이다!
  size_t number_of_blocks;
  number_of_blocks = (N + threads_per_block - 1) / threads_per_block;

  checkCuda(hipMallocManaged(&a, size));
  checkCuda(hipMallocManaged(&b, size));
  checkCuda(hipMallocManaged(&c, size));
  mallocCheck = hipGetLastError();
  if(mallocCheck != hipSuccess)
  {
    printf("Malloc failed");
    return -1;
  }
  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  addVectorsInto<<<number_of_blocks, threads_per_block>>>(c, a, b, N);
  syncError = hipGetLastError();
  asyncError = hipDeviceSynchronize();
  if(syncError != hipSuccess){
    printf("Error: %s\n", hipGetErrorString(syncError));
  }
  if(asyncError != hipSuccess)
  {
    printf("Error: %s\n", hipGetErrorString(asyncError));
  }
  checkElementsAre(7, c, N);
  
  hipFree(a);
  hipFree(b);
  hipFree(c);
}
