
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Currently, `initializeElementsTo`, if executed in a thread whose
 * `i` is calculated to be greater than `N`, will try to access a value
 * outside the range of `a`. -> 주어진 execution configuration <<a,b>> 보다 더 많은 영역을 접근하려고 하면 애초에 맞지 않는다. -> 따라서 initializElementsTO에서 해당 idx 위치가 N보다 작으면 값을 할당해줘야 한다. 
 *
 * Refactor the kernel defintition to prevent our of range accesses.
 */

__global__ void initializeElementsTo(int initialValue, int *a, int N)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < N)
  {
    a[i] = initialValue;
  }
  
}

int main()
{
  /*
   * Do not modify `N`.
   */

  int N = 1000;

  int *a;
  size_t size = N * sizeof(int);

  hipMallocManaged(&a, size);

  /*
   * Assume we have reason to want the number of threads
   * fixed at `256`: do not modify `threads_per_block`.
   */

  size_t threads_per_block = 256;

  /*
   * Assign a value to `number_of_blocks` that will
   * allow for a working execution configuration given
   * the fixed values for `N` and `threads_per_block`.
   */

  // Ensure there are at least `N` threads in the grid, but only 1 block's worth extra
  size_t number_of_blocks = (N+ threads_per_block -1) / threads_per_block;

  int initialValue = 6;

  initializeElementsTo<<<number_of_blocks, threads_per_block>>>(initialValue, a, N);
  hipDeviceSynchronize();

  /*
   * Check to make sure all values in `a`, were initialized.
   */

  for (int i = 0; i < N; ++i)
  {
    if(a[i] != initialValue)
    {
      printf("FAILURE: target value: %d\t a[%d]: %d\n", initialValue, i, a[i]);
      exit(1);
    }
  }
  printf("SUCCESS!\n");

  hipFree(a); // host 와 device에서 모두 접근할 수 있는 a는 cudaFree를 통해서 free한다.
}